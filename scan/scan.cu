#include "hip/hip_runtime.h"
#include <stdio.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#include <driver_functions.h>

#include <thrust/scan.h>
#include <thrust/device_ptr.h>
#include <thrust/device_malloc.h>
#include <thrust/device_free.h>

#include "CycleTimer.h"

#define THREADS_PER_BLOCK 256


// helper function to round an integer up to the next power of 2
static inline int nextPow2(int n) {
    n--;
    n |= n >> 1;
    n |= n >> 2;
    n |= n >> 4;
    n |= n >> 8;
    n |= n >> 16;
    n++;
    return n;
}

// CUDA kernel for upsweep phase of exclusive scan
__global__ void upsweep_kernel(int* data, int N, int two_d) {
    int two_dplus1 = two_d * 2;
    int thread_id = blockIdx.x * blockDim.x + threadIdx.x;

    // Map thread_id to actual array index that needs processing
    int i = thread_id * two_dplus1;

    if (i < N) {
        data[i + two_dplus1 - 1] += data[i + two_d - 1];
    }
}

// CUDA kernel for downsweep phase of exclusive scan
__global__ void downsweep_kernel(int* data, int N, int two_d) {
    int two_dplus1 = two_d * 2;
    int thread_id = blockIdx.x * blockDim.x + threadIdx.x;

    // Map thread_id to actual array index that needs processing
    int i = thread_id * two_dplus1;

    if (i < N) {
        int temp = data[i + two_d - 1];
        data[i + two_d - 1] = data[i + two_dplus1 - 1];
        data[i + two_dplus1 - 1] += temp;
    }
}

// exclusive_scan --
//
// Implementation of an exclusive scan on global memory array `input`,
// with results placed in global memory `result`.
//
// N is the logical size of the input and output arrays, however
// students can assume that both the start and result arrays we
// allocated with next power-of-two sizes as described by the comments
// in cudaScan().  This is helpful, since your parallel scan
// will likely write to memory locations beyond N, but of course not
// greater than N rounded up to the next power of 2.
//
// Also, as per the comments in cudaScan(), you can implement an
// "in-place" scan, since the timing harness makes a copy of input and
// places it in result
void exclusive_scan(int* input, int N, int* result)
{

    // CS149 TODO:
    //
    // Implement your exclusive scan implementation here.  Keep in
    // mind that although the arguments to this function are device
    // allocated arrays, this is a function that is running in a thread
    // on the CPU.  Your implementation will need to make multiple calls
    // to CUDA kernel functions (that you must write) to implement the
    // scan.

    int rounded_N = nextPow2(N);

    hipMemcpy(result, input, N * sizeof(int), hipMemcpyDeviceToDevice);

    // Upsweep phase
    for (int two_d = 1; two_d <= rounded_N/2; two_d *= 2) {
        int two_dplus1 = two_d * 2;

        // Calculate number of threads needed for this iteration
        int num_active_threads = rounded_N / two_dplus1;

        if (num_active_threads > 0) {
            int threads_per_block = min(THREADS_PER_BLOCK, num_active_threads);
            int num_blocks = (num_active_threads + threads_per_block - 1) / threads_per_block;

            upsweep_kernel<<<num_blocks, threads_per_block>>>(result, rounded_N, two_d);
            hipDeviceSynchronize();
        }
    }

    // Set the last element to zero for exclusive scan
    int zero = 0;
    hipMemcpy(&result[rounded_N - 1], &zero, sizeof(int), hipMemcpyHostToDevice);

    // Downsweep phase
    for (int two_d = rounded_N/2; two_d >= 1; two_d /= 2) {
        int two_dplus1 = 2 * two_d;
        
        // Calculate number of threads actually needed for this iteration
        int num_active_threads = rounded_N / two_dplus1;

        if (num_active_threads > 0) {
            // Launch only the threads we need
            int threads_per_block = min(num_active_threads, 512);
            int num_blocks = (num_active_threads + threads_per_block - 1) / threads_per_block;
            
            downsweep_kernel<<<num_blocks, threads_per_block>>>(result, rounded_N, two_d);
            hipDeviceSynchronize();
        }
    }   
}


//
// cudaScan --
//
// This function is a timing wrapper around the student's
// implementation of scan - it copies the input to the GPU
// and times the invocation of the exclusive_scan() function
// above. Students should not modify it.
double cudaScan(int* inarray, int* end, int* resultarray)
{
    int* device_result;
    int* device_input;
    int N = end - inarray;  

    // This code rounds the arrays provided to exclusive_scan up
    // to a power of 2, but elements after the end of the original
    // input are left uninitialized and not checked for correctness.
    //
    // Student implementations of exclusive_scan may assume an array's
    // allocated length is a power of 2 for simplicity. This will
    // result in extra work on non-power-of-2 inputs, but it's worth
    // the simplicity of a power of two only solution.

    int rounded_length = nextPow2(end - inarray);
    
    hipMalloc((void **)&device_result, sizeof(int) * rounded_length);
    hipMalloc((void **)&device_input, sizeof(int) * rounded_length);

    // For convenience, both the input and output vectors on the
    // device are initialized to the input values. This means that
    // students are free to implement an in-place scan on the result
    // vector if desired.  If you do this, you will need to keep this
    // in mind when calling exclusive_scan from find_repeats.
    hipMemcpy(device_input, inarray, (end - inarray) * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(device_result, inarray, (end - inarray) * sizeof(int), hipMemcpyHostToDevice);

    double startTime = CycleTimer::currentSeconds();

    exclusive_scan(device_input, N, device_result);

    // Wait for completion
    hipDeviceSynchronize();
    double endTime = CycleTimer::currentSeconds();
       
    hipMemcpy(resultarray, device_result, (end - inarray) * sizeof(int), hipMemcpyDeviceToHost);

    double overallDuration = endTime - startTime;
    return overallDuration; 
}


// cudaScanThrust --
//
// Wrapper around the Thrust library's exclusive scan function
// As above in cudaScan(), this function copies the input to the GPU
// and times only the execution of the scan itself.
//
// Students are not expected to produce implementations that achieve
// performance that is competition to the Thrust version, but it is fun to try.
double cudaScanThrust(int* inarray, int* end, int* resultarray) {

    int length = end - inarray;
    thrust::device_ptr<int> d_input = thrust::device_malloc<int>(length);
    thrust::device_ptr<int> d_output = thrust::device_malloc<int>(length);
    
    hipMemcpy(d_input.get(), inarray, length * sizeof(int), hipMemcpyHostToDevice);

    double startTime = CycleTimer::currentSeconds();

    thrust::exclusive_scan(d_input, d_input + length, d_output);

    hipDeviceSynchronize();
    double endTime = CycleTimer::currentSeconds();
   
    hipMemcpy(resultarray, d_output.get(), length * sizeof(int), hipMemcpyDeviceToHost);

    thrust::device_free(d_input);
    thrust::device_free(d_output);

    double overallDuration = endTime - startTime;
    return overallDuration; 
}

// CUDA kernel to creat flags array marking where A[i] == A[i+1]
__global__ void create_flags_kernel(int* input, int* flags, int length) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    if (i < length - 1) {
        flags[i] = (input[i] == input[i + 1]) ? 1 : 0;
    } else if (i == length - 1) {
        flags[i] = 0; // last element has no next element to compare
    }
}

// CUDA kernel to compact results using exclusive scan output
__global__ void compact_results_kernel(int* flags, int* scan_result, int* output, int length) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    if (i < length - 1 && flags[i] == 1) {
        output[scan_result[i]] = i; // store index of repeat
    }
}


// find_repeats --
//
// Given an array of integers `device_input`, returns an array of all
// indices `i` for which `device_input[i] == device_input[i+1]`.
//
// Returns the total number of pairs found
int find_repeats(int* device_input, int length, int* device_output) {

    // CS149 TODO:
    //
    // Implement this function. You will probably want to
    // make use of one or more calls to exclusive_scan(), as well as
    // additional CUDA kernel launches.
    //    
    // Note: As in the scan code, the calling code ensures that
    // allocated arrays are a power of 2 in size, so you can use your
    // exclusive_scan function with them. However, your implementation
    // must ensure that the results of find_repeats are correct given
    // the actual array length.

    int rounded_length = nextPow2(length);

    int* device_flags;
    int* device_scan_result;

    hipMalloc(&device_flags, rounded_length * sizeof(int));
    hipMalloc(&device_scan_result, rounded_length * sizeof(int));

    // Initialize arrays to 0 
    hipMemset(device_flags, 0, rounded_length * sizeof(int));
    hipMemset(device_scan_result, 0, rounded_length * sizeof(int));

    // Step 1: Create flags array
    int num_blocks = (length + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK;
    create_flags_kernel<<<num_blocks, THREADS_PER_BLOCK>>>(device_input, device_flags, length);
    hipDeviceSynchronize();

    // Step 2: Perform exclusive scan on flags array
    exclusive_scan(device_flags, length, device_scan_result);

    // Step 3: Compact results into output array
    compact_results_kernel<<<num_blocks, THREADS_PER_BLOCK>>>(device_flags, device_scan_result, device_output, length);
    hipDeviceSynchronize();

    // Step 4: Get the total count 
    int last_scan_value, last_flag_value;
    hipMemcpy(&last_scan_value, &device_scan_result[length - 1], sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(&last_flag_value, &device_flags[length - 1], sizeof(int), hipMemcpyDeviceToHost);

    int total_repeats = last_scan_value + last_flag_value;

    hipFree(device_flags);
    hipFree(device_scan_result);

    return total_repeats; // return the number of pairs found
}


//
// cudaFindRepeats --
//
// Timing wrapper around find_repeats. You should not modify this function.
double cudaFindRepeats(int *input, int length, int *output, int *output_length) {

    int *device_input;
    int *device_output;
    int rounded_length = nextPow2(length);
    
    hipMalloc((void **)&device_input, rounded_length * sizeof(int));
    hipMalloc((void **)&device_output, rounded_length * sizeof(int));
    hipMemcpy(device_input, input, length * sizeof(int), hipMemcpyHostToDevice);

    hipDeviceSynchronize();
    double startTime = CycleTimer::currentSeconds();
    
    int result = find_repeats(device_input, length, device_output);

    hipDeviceSynchronize();
    double endTime = CycleTimer::currentSeconds();

    // set output count and results array
    *output_length = result;
    hipMemcpy(output, device_output, length * sizeof(int), hipMemcpyDeviceToHost);

    hipFree(device_input);
    hipFree(device_output);

    float duration = endTime - startTime; 
    return duration;
}



void printCudaInfo()
{
    int deviceCount = 0;
    hipError_t err = hipGetDeviceCount(&deviceCount);

    printf("---------------------------------------------------------\n");
    printf("Found %d CUDA devices\n", deviceCount);

    for (int i=0; i<deviceCount; i++)
    {
        hipDeviceProp_t deviceProps;
        hipGetDeviceProperties(&deviceProps, i);
        printf("Device %d: %s\n", i, deviceProps.name);
        printf("   SMs:        %d\n", deviceProps.multiProcessorCount);
        printf("   Global mem: %.0f MB\n",
               static_cast<float>(deviceProps.totalGlobalMem) / (1024 * 1024));
        printf("   CUDA Cap:   %d.%d\n", deviceProps.major, deviceProps.minor);
    }
    printf("---------------------------------------------------------\n"); 
}
